#include "buildTensorflowGpu.h"

// Example of training a network on the buildTensorflow framework.
int main() {
    // Load Dataset
    Celsius2Faranheit<float,float> dataset;
    dataset.create(5);

    // Create Model
    Dense<float> fc1(1,1,NO_ACTIVATION);

    // Initialise Optimiser
    SGD<float> sgd(0.01);
    
    // Train
    cout<<"Training started"<<endl;
    for(int j = 0;j<2000;j++) {
        for(auto i: dataset.data) {
            // Get data
            auto inp = new Tensor<float>({i.first}, {1,1});
            auto tar = new Tensor<float>({i.second}, {1,1});

            // Forward Prop
            auto out = fc1.forward(inp);

            // Get Loss
            auto l = new Tensor<float>({-1}, {1,1});
            auto k = tensorOps::multiply(l,tar);
            auto loss = tensorOps::add(out,k); // error in loss
            auto finalLoss = tensorOps::power(loss,(float)2);

            // Compute backProp
            finalLoss->backward();
            // cout<<finalLoss->val<<endl;

            // Perform Gradient Descent
            sgd.minimise(finalLoss);
        
        }
    }

    cout<<"Training completed"<<endl;

    // Inference
    float cel = 4;
    auto test = new Tensor<float>({cel}, {1,1});
    auto out1 = fc1.forward(test);

    cout<<"The conversion of "<<cel<<" degrees celcius to faranheit is "<<out1->val<<endl; // For 4 Celcius: it's ~39.2

    // Clean up
    delete out1;
}

